#include "hip/hip_runtime.h"
#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

inline void checkCudaStatus(hipError_t status)
{
    if (status != hipSuccess)
    {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        throw std::logic_error("cuda API failed");
    }
}

inline void checkCublasStatus(hipblasStatus_t status)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("cuBLAS API failed with status %d\n", status);
        throw std::logic_error("cuBLAS API failed");
    }
}

torch::Tensor cublas_gemm_kernel(torch::Tensor a, torch::Tensor b, int m, int n, int k, bool trans_a = false, bool trans_b = false, int lda = -1, int ldb = -1, int ldc = -1, int out_h = -1, int out_w = -1)
{
    c10::cuda::CUDAGuard device_guard(a.device());

    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();

    at::Half alpha = 1.0f;
    at::Half beta = 0.0f;

    if (out_h == -1)
    {
        out_h = m;
    }

    if (out_w == -1)
    {
        out_w = n;
    }

    torch::Tensor out = torch::empty({out_h, out_w}, a.options().device(a.device()));

    hipblasOperation_t OP_A = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t OP_B = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    if (lda == -1)
    {
        lda = OP_A == HIPBLAS_OP_N ? m : k;
    }
    if (ldb == -1)
    {
        ldb = OP_B == HIPBLAS_OP_N ? k : n;
    }
    if (ldc == -1)
    {
        ldc = m;
    }
    hipblasStatus_t result = hipblasHgemm(
        handle,
        OP_A,
        OP_B,
        m,
        n,
        k,
        (__half *)&alpha,
        (__half *)a.const_data_ptr<at::Half>(),
        lda,
        (__half *)b.const_data_ptr<at::Half>(),
        ldb,
        (__half *)&beta,
        (__half *)out.mutable_data_ptr<at::Half>(),
        ldc);
    if (result != HIPBLAS_STATUS_SUCCESS)
    {
        const char *results = hipblasStatusToString(result);

        std::string error_message = "hipblasGemmEx failed with error: ";
        error_message += results;

        std::cout << error_message << std::endl;

        throw std::runtime_error(results);
    }

    return out;
}

// b is equal to A, and a is equal to B, because cublas is opposite land.
torch::Tensor cublas_gemm_kernel_axbT(torch::Tensor a, torch::Tensor b)
{
    at::Half alpha = 1.0f;
    at::Half beta = 0.0f;

    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();

    int m = a.size(0);
    int n = b.size(0);
    int k = a.size(1);

    int lda = a.size(1);
    int ldb = b.size(1);

    int ldc = m;
    int out_h = n;
    int out_w = m;

    // output as (N, M) - because opposite land.
    torch::Tensor out = torch::empty({out_h, out_w}, a.options().device(a.device()));

    /*
    ### REFERENCE ###
    hipblasStatus_t hipblasHgemm(
        hipblasHandle_t handle,
        hipblasOperation_t transa,
        hipblasOperation_t transb,
        int m,
        int n,
        int k,
        const half *alpha,
        const half *A,
        int lda,
        const half *B,
        int ldb,
        const half *beta,
        half *C,
        int ldc
    );
    */
    checkCublasStatus(hipblasHgemm(
        handle,
        // Transpose a
        HIPBLAS_OP_T,
        // Don't transpose b
        HIPBLAS_OP_N,
        m,
        n,
        k,
        (__half *)&alpha,
        (__half *)a.const_data_ptr<at::Half>(),
        lda,
        (__half *)b.const_data_ptr<at::Half>(),
        ldb,
        (__half *)&beta,
        (__half *)out.mutable_data_ptr<at::Half>(),
        ldc));

    return out;
}