#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAContextLight.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#define DEFAULT_WORKSPACE_SIZE 134217728

inline void checkCudaStatus(hipError_t status)
{
    if (status != hipSuccess)
    {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        throw std::logic_error("cuda API failed");
    }
}

inline void checkCublasStatus(hipblasStatus_t status)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("cuBLAS API failed with status %d\n", status);
        throw std::logic_error("cuBLAS API failed");
    }
}

torch::Tensor cublaslt_gemm_launch_axbT(
    torch::Tensor a,
    torch::Tensor b,
    torch::Tensor bias = {},
    std::string epilogue_str = "NONE",
    torch::Tensor workspace = {},
    size_t workspaceSize = DEFAULT_WORKSPACE_SIZE,
    bool trans_a = false,
    bool trans_b = false,
    int m = -1,
    int n = -1,
    int k = -1,
    int lda = -1,
    int ldb = -1,
    int ldc = -1,
    int out_h = -1,
    int out_w = -1)
{
    c10::cuda::CUDAGuard device_guard(a.device());

    hipblasLtHandle_t ltHandle = at::cuda::getCurrentCUDABlasLtHandle();
    bool has_bias = bias.numel() > 0;

    if (workspace.numel() == 0)
    {
        // Allocate workspace if not provided
        workspace = torch::empty(workspaceSize, at::TensorOptions().dtype(torch::kUInt8).device(a.device()));
    }

    // Do confusing stuff with dimensions because magic happens.

    m = m == -1 ? a.size(0) : m;
    n = n == -1 ? b.size(0) : n;
    k = k == -1 ? a.size(1) : k;
    lda = lda == -1 ? a.size(1) : lda;
    ldb = ldb == -1 ? b.size(1) : ldb;
    ldc = ldc == -1 ? m : ldc;
    out_h = out_h == -1 ? n : out_h;
    out_w = out_w == -1 ? m : out_w;

    // Get bias pointer and data type
    // Will only be used if has_bias is true
    half *bias_ref = (half *)bias.const_data_ptr<at::Half>();
    hipDataType cublasBiasDataType = HIP_R_16F;

    // Allocate output tensor
    torch::Tensor out = torch::empty({out_h, out_w}, a.options().device(a.device()));

    // Create hipblasLtMatmulDesc_t's, hipblasLtMatrixLayout_t's, and hipblasLtMatmulPreference_t
    hipblasLtMatmulDesc_t operationDesc = nullptr;
    hipblasLtMatrixLayout_t aDesc = nullptr, bDesc = nullptr, outDesc = nullptr;
    hipblasLtMatmulPreference_t preference = nullptr;

    // Handle transposition
    hipblasOperation_t transa = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transb = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    // Create matmul operation descriptor with fp16 accumulation
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_16F, HIP_R_16F));

    // Set transposition attributes
    checkCublasStatus(
        hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(
        hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // Parse epilogue type
    hipblasLtEpilogue_t epilogue;

    // Set epilogue attributes
    if (epilogue_str == "NONE")
    {

        epilogue = has_bias ? HIPBLASLT_EPILOGUE_BIAS : HIPBLASLT_EPILOGUE_DEFAULT;
    }
    else if (epilogue_str == "RELU")
    {

        epilogue = has_bias ? HIPBLASLT_EPILOGUE_RELU_BIAS : HIPBLASLT_EPILOGUE_RELU;
    }
    else if (epilogue_str == "GELU")
    {

        epilogue = has_bias ? HIPBLASLT_EPILOGUE_GELU_BIAS : HIPBLASLT_EPILOGUE_GELU;
    }
    else
    {

        throw std::invalid_argument("Invalid epilogue type");
    }
    // Set epilogue attribute in operation descriptor
    checkCublasStatus(
        hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));

    // Set bias attributes if bias is provided
    if (has_bias)
    {
        checkCublasStatus(hipblasLtMatmulDescSetAttribute(
            operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias_ref, sizeof(bias_ref)));
        checkCublasStatus(hipblasLtMatmulDescSetAttribute(
            operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_DATA_TYPE, &cublasBiasDataType, sizeof(cublasBiasDataType)));
    }

    // Create matrix layout descriptors based on input tensors shapes
    checkCublasStatus(hipblasLtMatrixLayoutCreate(
        &aDesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(
        &bDesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&outDesc, HIP_R_16F, m, n, ldc));

    // Create preference descriptor & set workspace size
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    /*

    ### FOR REFERENCE ###

    hipblasStatus_t hipblasLtMatmul(
        hipblasLtHandle_t lightHandle,
        hipblasLtMatmulDesc_t computeDesc,
        const void *alpha, const void *A,
        hipblasLtMatrixLayout_t Adesc,
        const void *B,
        hipblasLtMatrixLayout_t Bdesc,
        const void *beta,
        const void *C,
        hipblasLtMatrixLayout_t Cdesc,
        void *D,
        hipblasLtMatrixLayout_t Ddesc,
        const hipblasLtMatmulAlgo_t *algo,
        void *workspace,
        size_t workspaceSizeInBytes,
        hipStream_t stream
    )
    */
    const at::Half alpha = 1.0f;
    const at::Half beta = 0.0f;

    // CUDA GO NYOOM NYOOM
    checkCublasStatus(hipblasLtMatmul(
        ltHandle,
        operationDesc,
        (const void *)&alpha,
        (const void *)a.const_data_ptr<at::Half>(),
        aDesc,
        (const void *)b.const_data_ptr<at::Half>(),
        bDesc,
        (const void *)&beta,
        (const void *)out.mutable_data_ptr<at::Half>(),
        outDesc,
        (void *)out.mutable_data_ptr<at::Half>(),
        outDesc,
        nullptr,
        (void *)workspace.mutable_data_ptr<uint8_t>(),
        workspaceSize,
        at::cuda::getCurrentCUDAStream(a.device().index())));

    // Clean up
    checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(outDesc));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(bDesc));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(aDesc));
    checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));

    return out;
}

torch::Tensor cublaslt_hgemm_customizable(
    torch::Tensor a,
    torch::Tensor b,
    torch::Tensor bias = {},
    std::string epilogue_str = "NONE",
    torch::Tensor workspace = {},
    bool trans_a = false,
    bool trans_b = false,
    int m = -1,
    int n = -1,
    int k = -1,
    int lda = -1,
    int ldb = -1,
    int ldc = -1,
    int out_h = -1,
    int out_w = -1,
    size_t workspaceSize = DEFAULT_WORKSPACE_SIZE)
{
    return cublaslt_gemm_launch_axbT(
        a, b, bias, epilogue_str, workspace, workspaceSize, trans_a, trans_b, m, n, k, lda, ldb, ldc, out_h, out_w);
}

torch::Tensor cublaslt_hgemm_simple(
    torch::Tensor a,
    torch::Tensor b,
    torch::Tensor bias = {},
    std::string epilogue_str = "NONE",
    torch::Tensor workspace = {})
{
    size_t workspace_sz = DEFAULT_WORKSPACE_SIZE;
    if (workspace.numel() > 0)
    {
        workspace_sz = workspace.numel();
    }
    return cublaslt_gemm_launch_axbT(
        b, a, bias, epilogue_str, workspace, workspace_sz, true, false);
}
